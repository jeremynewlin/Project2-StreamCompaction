#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <iostream>

#include "streamCompaction.h"

__global__ void sum(int* in, int* out, int n, int d1){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (k<n){
    int ink = in[k];
    if (k>=d1){
      out[k] = in[k-d1] + ink;
    }
    else{
      out[k] = ink;
    }
  }
}

__global__ void shift(int* in, int* out, int n){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  out[0] = 0;
  if (k<n && k>0){
    out[k] = in[k-1];
  }
}

__global__ void streamCompaction(dataPacket* inRays, int* indices, dataPacket* outRays, int numElements){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k<numElements){
    dataPacket inRay = inRays[k];
    if (inRay.alive){
      outRays[indices[k]-1] = inRay;
    }
  }
}

__global__ void killStream(int index, dataPacket* inRays, int* indices, int numElements){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k<numElements){
    inRays[k].alive = true;
    indices[k] = 1;
    if (k == index){
      inRays[k].alive = false;
      indices[k] = 0;
    }
  }
}

void testStreamCompaction(){
  //Testing stream compaction
  int numElements = 10;
  dataPacket* arrayOfElements = new dataPacket[numElements];
  for (int i=0; i<numElements; i+=1){
    dataPacket rb(i);
    arrayOfElements[i] = rb;
  }

  arrayOfElements[1].alive=false;
  arrayOfElements[4].alive=false;
  arrayOfElements[5].alive=false;
  arrayOfElements[7].alive=false;
  arrayOfElements[8].alive=false;


  dataPacket* cudaArrayA;
  dataPacket* cudaArrayB;

  hipMalloc((void**)&cudaArrayA, numElements*sizeof(dataPacket));
  hipMalloc((void**)&cudaArrayB, numElements*sizeof(dataPacket));

  int* testin;
  int* testout;
  int* cputest = new int[numElements];

  for (int i=0; i<numElements; i++){
    if (arrayOfElements[i].alive){
      cputest[i]=1;
    }
    else{
      cputest[i]=0;
    }
  }

  hipMalloc((void**)&testin, numElements*sizeof(int));
  hipMalloc((void**)&testout, numElements*sizeof(int));

  hipMemcpy(cudaArrayA, arrayOfElements, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);  
  hipMemcpy(cudaArrayB, arrayOfElements, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);  
  hipMemcpy(testin, cputest, numElements*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(testout, cputest, numElements*sizeof(int), hipMemcpyHostToDevice);

  for (int i=0; i<numElements; i++){
    std::cout<<arrayOfElements[i].index<<", "<<cputest[i]<<std::endl;
  }

  dim3 threadsPerBlock(64);
  dim3 fullBlocksPerGrid(int(ceil(float(numElements)/64.0f)));

  //scan
  for (int d=1; d<=ceil(log(numElements)/log(2))+1; d++){
    sum<<<fullBlocksPerGrid, threadsPerBlock>>>(testin, testout, numElements, int(pow(2.0f,d-1)));
    hipDeviceSynchronize();
    hipMemcpy(cputest, testout, numElements*sizeof(int), hipMemcpyDeviceToHost);


    int* temp = testin;
    testin=testout;
    testout=temp;
  }
  //Compact
  streamCompaction<<<fullBlocksPerGrid, threadsPerBlock>>>(cudaArrayA, testin, cudaArrayB, numElements);
  cudaArrayA = cudaArrayB;
  hipDeviceSynchronize();

  hipMemcpy(&numElements, &testin[numElements-1], 1*sizeof(int), hipMemcpyDeviceToHost);

  std::cout<<"number of rays left: "<<numElements<<std::endl;

  // for (int i=0; i<numElements; i++){
  //   std::cout<<cputest[i]<<std::endl;
  // }    

  hipMemcpy(cputest, testin, numElements*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(arrayOfElements, cudaArrayA, numElements*sizeof(dataPacket), hipMemcpyDeviceToHost);


  for (int i=0; i<numElements; i++){
    std::cout<<arrayOfElements[i].index<<std::endl;

  }
  std::cout<<"___________________________________"<<std::endl;


  delete [] cputest;
  hipFree(testin);
  hipFree(testout);

  delete [] arrayOfElements;
  hipFree(cudaArrayA);
  hipFree(cudaArrayB);
}

DataStream::DataStream(int numElements, dataPacket * data){
  m_data = data;
  m_numElementsAlive = numElements;
  m_numElements = numElements;

  m_indices = new int[numElements];
  for (int i=0; i<numElements; i+=1){
    m_indices[i] = 1;
  }

  //cudaInit (cudaDataA, cudaDataB, cudaIndicesA, cudaIndicesB);
  hipMalloc ((void**)&cudaDataA, numElements*sizeof (dataPacket));
  hipMalloc ((void**)&cudaDataB, numElements*sizeof (dataPacket));
  hipMalloc ((void**)&cudaIndicesA, numElements*sizeof (int));
  hipMalloc ((void**)&cudaIndicesB, numElements*sizeof (int));

  hipMemcpy(cudaDataA, m_data, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);
  hipMemcpy(cudaDataB, m_data, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);
  hipMemcpy(cudaIndicesA, m_indices, numElements*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaIndicesB, m_indices, numElements*sizeof(int), hipMemcpyHostToDevice);
}

DataStream::~DataStream(){
  hipFree (cudaDataA);
  hipFree (cudaDataB);
  hipFree (cudaIndicesA);
  hipFree (cudaIndicesB);

  delete [] m_data;
  delete [] m_indices;
}

void DataStream::compact(){
  dim3 threadsPerBlockL(64);
  dim3 fullBlocksPerGridL(int(ceil(float(m_numElementsAlive)/64.0f)));

  // scan algorithm
    for (int d=1; d<=ceil(log(m_numElementsAlive)/log(2)); d++){
      sum<<<fullBlocksPerGridL, threadsPerBlockL>>>(cudaIndicesA, cudaIndicesB, m_numElementsAlive, powf(2.0f, d-1));
      int* temp = cudaIndicesA;
      cudaIndicesA = cudaIndicesB;
      cudaIndicesB = temp;
    }

    //Stream compation from A into B, then save back into A
    streamCompaction<<<fullBlocksPerGridL, threadsPerBlockL>>>(cudaDataA, cudaIndicesA, cudaDataB, m_numElementsAlive);
    dataPacket * temp = cudaDataA;
    cudaDataA = cudaDataB;
    cudaDataB = temp;

    // update numrays
    hipMemcpy(&m_numElementsAlive, &cudaIndicesA[m_numElementsAlive-1], sizeof(int), hipMemcpyDeviceToHost);
}

bool DataStream::getData(int index, dataPacket& data){

  if (index > m_numElements) return false;

  data = m_data[index];
  return true;
}

int DataStream::numAlive(){
  return m_numElementsAlive;
}

void DataStream::fetchDataFromGPU(){
  hipMemcpy(m_data, cudaDataA, m_numElementsAlive*sizeof(dataPacket), hipMemcpyDeviceToHost);
}

void DataStream::kill(int index){
  if (index > m_numElementsAlive) return;

  dim3 threadsPerBlockL(64);
  dim3 fullBlocksPerGridL(int(ceil(float(m_numElementsAlive)/64.0f)));

  killStream<<<fullBlocksPerGridL, threadsPerBlockL>>>(index, cudaDataA, cudaIndicesA, m_numElementsAlive);
}