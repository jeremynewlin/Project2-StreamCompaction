#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <iostream>

#include "streamCompaction.h"

using namespace std;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
  }
} 

__global__ void sum(int* in, int* out, int n, int d1){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (k<n){
    int ink = in[k];
    if (k>=d1){
      out[k] = in[k-d1] + ink;
    }
    else{
      out[k] = ink;
    }
  }
}

__global__ void test(int* in, int* out, int n){

  extern __shared__ float temp[];
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int offset = 1;

  if (2*index+1<=n){
    temp[2*index] = in[2*index];
    temp[2*index+1] = in[2*index+1];

    for (int d = n>>1; d>0; d >>= 1){
    //for (int d=0; d<floor(log(float(n-1)/log(2.0f))); d+=1){
      __syncthreads();
      if (index < d){
        int ai = offset * (2*index+1) - 1;
        int bi = offset * (2*index+2) - 1;

        temp[bi] += temp[ai];
      }
      offset *= 2;
    }

    if (index == 0) temp[n - 1] = 0;

    for (int d = 1; d<n; d*=2){
      offset >>= 1;
      __syncthreads();
      if (index < d){

        int ai = offset * (2*index+1) - 1;
        int bi = offset * (2*index+2) - 1;

        if (ai < n && bi < n){
          float t = temp[ai];
          temp[ai] = temp[bi];
          temp[bi] += t;
        }
      }
    }
    __syncthreads();

    out[2*index] = temp[2*index];
    out[2*index+1] = temp[2*index+1];
  }

}

__global__ void test2(int* in, int* out, int n, int* sums=0){

  extern __shared__ float temp[];

  int realIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  temp[realIndex] = 0;
  
  int offset = 1;
  int index = threadIdx.x;

  temp[2*index] = in[2*realIndex];
  temp[2*index+1] = in[2*realIndex+1];

  for (int d = n>>1; d>0; d >>= 1){
    __syncthreads();
    if (index < d){
      int ai = offset * (2*index+1) - 1;
      int bi = offset * (2*index+2) - 1;

      temp[bi] += temp[ai];
    }
    offset *= 2;
  }
  
  if (index == 0){
    if (sums) sums[blockIdx.x] = temp[16-1];
    temp[n - 1] = 0;
  }

  for (int d = 1; d<n; d*=2){
    offset >>= 1;
    __syncthreads();
    if (index < d){

      int ai = offset * (2*index+1) - 1;
      int bi = offset * (2*index+2) - 1;

      if (ai < n && bi < n){
        float t = temp[ai];
        temp[ai] = temp[bi];
        temp[bi] += t;
      }
    }
  }
  __syncthreads();

  out[2*realIndex] = temp[2*index];
  out[2*realIndex+1] = temp[2*index+1];

}

__global__ void addIncs(int* cudaAuxIncs, int* cudaIndicesB, int n){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;


  // if (index < n){
    // cudaIndicesB[index] = blockIdx.x; //cudaAuxIncs[blockIdx.x];
    cudaIndicesB[index] += cudaAuxIncs[blockIdx.x];
  // }
}

__global__ void streamCompaction(dataPacket* inRays, int* indices, dataPacket* outRays, int numElements){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k<numElements){
    dataPacket inRay = inRays[k];
    if (inRay.alive){
      outRays[indices[k]-1] = inRay;
    }
  }
}

__global__ void killStream(int index, dataPacket* inRays, int* indices, int numElements){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k<numElements){
    inRays[k].alive = true;
    indices[k] = 1;
    if (k == index){
      inRays[k].alive = false;
      indices[k] = 0;
    }
  }
}

void testStreamCompaction(){
  //Testing stream compaction
  int numElements = 10;
  dataPacket* arrayOfElements = new dataPacket[numElements];
  for (int i=0; i<numElements; i+=1){
    dataPacket rb(i);
    arrayOfElements[i] = rb;
  }

  arrayOfElements[1].alive=false;
  arrayOfElements[4].alive=false;
  arrayOfElements[5].alive=false;
  arrayOfElements[7].alive=false;
  arrayOfElements[8].alive=false;


  dataPacket* cudaArrayA;
  dataPacket* cudaArrayB;

  hipMalloc((void**)&cudaArrayA, numElements*sizeof(dataPacket));
  hipMalloc((void**)&cudaArrayB, numElements*sizeof(dataPacket));

  int* testin;
  int* testout;
  int* cputest = new int[numElements];

  for (int i=0; i<numElements; i++){
    if (arrayOfElements[i].alive){
      cputest[i]=1;
    }
    else{
      cputest[i]=0;
    }
  }

  hipMalloc((void**)&testin, numElements*sizeof(int));
  hipMalloc((void**)&testout, numElements*sizeof(int));

  hipMemcpy(cudaArrayA, arrayOfElements, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);  
  hipMemcpy(cudaArrayB, arrayOfElements, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);  
  hipMemcpy(testin, cputest, numElements*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(testout, cputest, numElements*sizeof(int), hipMemcpyHostToDevice);

  for (int i=0; i<numElements; i++){
    std::cout<<arrayOfElements[i].index<<", "<<cputest[i]<<std::endl;
  }

  dim3 threadsPerBlock(64);
  dim3 fullBlocksPerGrid(int(ceil(float(numElements)/64.0f)));

  //scan
  for (int d=1; d<=ceil(log(numElements)/log(2))+1; d++){
    sum<<<fullBlocksPerGrid, threadsPerBlock>>>(testin, testout, numElements, int(pow(2.0f,d-1)));
    hipDeviceSynchronize();
    hipMemcpy(cputest, testout, numElements*sizeof(int), hipMemcpyDeviceToHost);


    int* temp = testin;
    testin=testout;
    testout=temp;
  }
  //Compact
  streamCompaction<<<fullBlocksPerGrid, threadsPerBlock>>>(cudaArrayA, testin, cudaArrayB, numElements);
  cudaArrayA = cudaArrayB;
  hipDeviceSynchronize();

  hipMemcpy(&numElements, &testin[numElements-1], 1*sizeof(int), hipMemcpyDeviceToHost);

  std::cout<<"number of rays left: "<<numElements<<std::endl;

  // for (int i=0; i<numElements; i++){
  //   std::cout<<cputest[i]<<std::endl;
  // }    

  hipMemcpy(cputest, testin, numElements*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(arrayOfElements, cudaArrayA, numElements*sizeof(dataPacket), hipMemcpyDeviceToHost);


  for (int i=0; i<numElements; i++){
    std::cout<<arrayOfElements[i].index<<std::endl;

  }
  std::cout<<"___________________________________"<<std::endl;


  delete [] cputest;
  hipFree(testin);
  hipFree(testout);

  delete [] arrayOfElements;
  hipFree(cudaArrayA);
  hipFree(cudaArrayB);
}

DataStream::DataStream(int numElements, dataPacket * data){
  m_data = data;
  m_numElementsAlive = numElements;

  // if (numElements % THREADS_PER_BLOCK*2 != 0){
  //   int counter = 1;
  //   while (THREADS_PER_BLOCK*2*counter < numElements){
  //     counter += 1;
  //   }
  //   numElements = THREADS_PER_BLOCK*2*counter;
  // }

  // cout<<numElements<<endl;

  m_numElements = numElements;

  m_indices = new int[numElements];
  for (int i=0; i<numElements; i+=1){
    if (i < m_numElementsAlive){
      m_indices[i] = 1;
    }
    else{
      m_indices[i] = 0;
    }
  }

  m_auxSums = new int[numElements/(THREADS_PER_BLOCK*2)];
  for (int i=0; i<numElements/(THREADS_PER_BLOCK*2); i+=1){
    m_auxSums[i] = 0;
  }

  //cudaInit (cudaDataA, cudaDataB, cudaIndicesA, cudaIndicesB);
  hipMalloc ((void**)&cudaDataA, numElements*sizeof (dataPacket));
  hipMalloc ((void**)&cudaDataB, numElements*sizeof (dataPacket));
  hipMalloc ((void**)&cudaIndicesA, numElements*sizeof (int));
  hipMalloc ((void**)&cudaIndicesB, numElements*sizeof (int));
  hipMalloc ((void**)&cudaAuxSums, numElements/(THREADS_PER_BLOCK*2)*sizeof (int));
  hipMalloc ((void**)&cudaAuxIncs, numElements/(THREADS_PER_BLOCK*2)*sizeof (int));

  hipMemcpy(cudaDataA, m_data, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);
  hipMemcpy(cudaDataB, m_data, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);
  hipMemcpy(cudaIndicesA, m_indices, numElements*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaIndicesB, m_indices, numElements*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaAuxSums, m_auxSums, numElements/(THREADS_PER_BLOCK*2)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaAuxIncs, m_auxSums, numElements/(THREADS_PER_BLOCK*2)*sizeof(int), hipMemcpyHostToDevice);
}

DataStream::~DataStream(){
  hipFree (cudaDataA);
  hipFree (cudaDataB);
  hipFree (cudaIndicesA);
  hipFree (cudaIndicesB);
  hipFree (cudaAuxSums);
  hipFree (cudaAuxIncs);

  delete [] m_data;
  delete [] m_indices;
  delete [] m_auxSums;
}

void DataStream::compact(){

  int numElements = m_numElementsAlive;
  int threadsPerBlock = THREADS_PER_BLOCK; // 8
  int procsPefBlock = threadsPerBlock*2;   // 16

  dim3 initialScanThreadsPerBlock(procsPefBlock/2);        //8
  dim3 initialScanBlocksPerGrid(numElements/procsPefBlock);//

  int sumSize = numElements/(THREADS_PER_BLOCK*2);

  dim3 initialScanThreadsPerBlock2(sumSize/2);        //16
  dim3 initialScanBlocksPerGrid2(sumSize/(sumSize/2)+1);//1024/16

  dim3 initialScanThreadsPerBlock3(procsPefBlock);        //8
  dim3 initialScanBlocksPerGrid3(numElements/procsPefBlock);//3

  dim3 threadsPerBlockL(threadsPerBlock);
  dim3 fullBlocksPerGridL(int(ceil(float(m_numElementsAlive)/float(threadsPerBlock))));

  test2<<<initialScanBlocksPerGrid, initialScanThreadsPerBlock, m_numElements*sizeof(int)>>>(cudaIndicesA, cudaIndicesB, procsPefBlock, cudaAuxSums);
  checkCUDAError("kernel failed!");

  test2<<<initialScanBlocksPerGrid2, initialScanThreadsPerBlock2, m_numElements*sizeof(int)>>>(cudaAuxSums, cudaAuxIncs, sumSize);
  checkCUDAError("kernel failed!");

  addIncs<<<initialScanBlocksPerGrid3, initialScanThreadsPerBlock3>>>(cudaAuxIncs, cudaIndicesB, m_numElements);
  checkCUDAError("kernel failed!");

  hipMemcpy(m_indices, cudaIndicesB, m_numElements*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(m_auxSums, cudaAuxIncs, m_numElements/(THREADS_PER_BLOCK*2)*sizeof(int), hipMemcpyDeviceToHost);

  // // scan algorithm
  //   for (int d=1; d<=ceil(log(m_numElementsAlive)/log(2)); d++){
  //     sum<<<fullBlocksPerGridL, threadsPerBlockL>>>(cudaIndicesA, cudaIndicesB, m_numElementsAlive, powf(2.0f, d-1));
  //     int* temp = cudaIndicesA;
  //     cudaIndicesA = cudaIndicesB;
  //     cudaIndicesB = temp;
  //   }

    // test<<<fullBlocksPerGridL, threadsPerBlockL, m_numElementsAlive*sizeof(int)>>>(cudaIndicesA, cudaIndicesB, m_numElementsAlive);
    // checkCUDAError("kernel failed!");
    // hipMemcpy(m_indices, cudaIndicesB, m_numElements*sizeof(int), hipMemcpyDeviceToHost);
    // //Stream compation from A into B, then save back into A
    // streamCompaction<<<fullBlocksPerGridL, threadsPerBlockL>>>(cudaDataA, cudaIndicesA, cudaDataB, m_numElementsAlive);
    // dataPacket * temp = cudaDataA;
    // cudaDataA = cudaDataB;
    // cudaDataB = temp;

    // // update numrays
    // hipMemcpy(&m_numElementsAlive, &cudaIndicesA[m_numElementsAlive-1], sizeof(int), hipMemcpyDeviceToHost);
}

bool DataStream::getData(int index, dataPacket& data){

  if (index > m_numElements) return false;

  data = m_data[index];
  return true;
}

int DataStream::numAlive(){
  return m_numElementsAlive;
}

void DataStream::fetchDataFromGPU(){
  hipMemcpy(m_data, cudaDataA, m_numElementsAlive*sizeof(dataPacket), hipMemcpyDeviceToHost);
}

void DataStream::kill(int index){
  if (index > m_numElementsAlive) return;

  dim3 threadsPerBlockL(64);
  dim3 fullBlocksPerGridL(int(ceil(float(m_numElementsAlive)/64.0f)));

  killStream<<<fullBlocksPerGridL, threadsPerBlockL>>>(index, cudaDataA, cudaIndicesA, m_numElementsAlive);
}