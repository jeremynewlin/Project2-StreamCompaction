#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <iostream>

#include "streamCompaction.h"

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
  }
} 

__global__ void sum(int* in, int* out, int n, int d1){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (k<n){
    int ink = in[k];
    if (k>=d1){
      out[k] = in[k-d1] + ink;
    }
    else{
      out[k] = ink;
    }
  }
}

__global__ void test(int* in, int* out, int n){

  extern __shared__ float temp[];
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int offset = 1;

  if (2*index+1<=n){
    temp[2*index] = in[2*index];
    temp[2*index+1] = in[2*index+1];

    for (int d = n>>1; d>0; d >>= 1){
    //for (int d=0; d<floor(log(float(n-1)/log(2.0f))); d+=1){
      __syncthreads();
      if (index < d){
        int ai = offset * (2*index+1) - 1;
        int bi = offset * (2*index+2) - 1;

        temp[bi] += temp[ai];
      }
      offset *= 2;
    }

    if (index == 0) temp[n - 1] = 0;

    for (int d = 1; d<n; d*=2){
      offset >>= 1;
      __syncthreads();
      if (index < d){

        int ai = offset * (2*index+1) - 1;
        int bi = offset * (2*index+2) - 1;

        if (ai < n && bi < n){
          float t = temp[ai];
          temp[ai] = temp[bi];
          temp[bi] += t;
        }
      }
    }
    __syncthreads();

    out[2*index] = temp[2*index];
    out[2*index+1] = temp[2*index+1];
  }

}

__global__ void streamCompaction(dataPacket* inRays, int* indices, dataPacket* outRays, int numElements){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k<numElements){
    dataPacket inRay = inRays[k];
    if (inRay.alive){
      outRays[indices[k]-1] = inRay;
    }
  }
}

__global__ void killStream(int index, dataPacket* inRays, int* indices, int numElements){
  int k = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (k<numElements){
    inRays[k].alive = true;
    indices[k] = 1;
    if (k == index){
      inRays[k].alive = false;
      indices[k] = 0;
    }
  }
}

void testStreamCompaction(){
  //Testing stream compaction
  int numElements = 10;
  dataPacket* arrayOfElements = new dataPacket[numElements];
  for (int i=0; i<numElements; i+=1){
    dataPacket rb(i);
    arrayOfElements[i] = rb;
  }

  arrayOfElements[1].alive=false;
  arrayOfElements[4].alive=false;
  arrayOfElements[5].alive=false;
  arrayOfElements[7].alive=false;
  arrayOfElements[8].alive=false;


  dataPacket* cudaArrayA;
  dataPacket* cudaArrayB;

  hipMalloc((void**)&cudaArrayA, numElements*sizeof(dataPacket));
  hipMalloc((void**)&cudaArrayB, numElements*sizeof(dataPacket));

  int* testin;
  int* testout;
  int* cputest = new int[numElements];

  for (int i=0; i<numElements; i++){
    if (arrayOfElements[i].alive){
      cputest[i]=1;
    }
    else{
      cputest[i]=0;
    }
  }

  hipMalloc((void**)&testin, numElements*sizeof(int));
  hipMalloc((void**)&testout, numElements*sizeof(int));

  hipMemcpy(cudaArrayA, arrayOfElements, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);  
  hipMemcpy(cudaArrayB, arrayOfElements, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);  
  hipMemcpy(testin, cputest, numElements*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(testout, cputest, numElements*sizeof(int), hipMemcpyHostToDevice);

  for (int i=0; i<numElements; i++){
    std::cout<<arrayOfElements[i].index<<", "<<cputest[i]<<std::endl;
  }

  dim3 threadsPerBlock(64);
  dim3 fullBlocksPerGrid(int(ceil(float(numElements)/64.0f)));

  //scan
  for (int d=1; d<=ceil(log(numElements)/log(2))+1; d++){
    sum<<<fullBlocksPerGrid, threadsPerBlock>>>(testin, testout, numElements, int(pow(2.0f,d-1)));
    hipDeviceSynchronize();
    hipMemcpy(cputest, testout, numElements*sizeof(int), hipMemcpyDeviceToHost);


    int* temp = testin;
    testin=testout;
    testout=temp;
  }
  //Compact
  streamCompaction<<<fullBlocksPerGrid, threadsPerBlock>>>(cudaArrayA, testin, cudaArrayB, numElements);
  cudaArrayA = cudaArrayB;
  hipDeviceSynchronize();

  hipMemcpy(&numElements, &testin[numElements-1], 1*sizeof(int), hipMemcpyDeviceToHost);

  std::cout<<"number of rays left: "<<numElements<<std::endl;

  // for (int i=0; i<numElements; i++){
  //   std::cout<<cputest[i]<<std::endl;
  // }    

  hipMemcpy(cputest, testin, numElements*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(arrayOfElements, cudaArrayA, numElements*sizeof(dataPacket), hipMemcpyDeviceToHost);


  for (int i=0; i<numElements; i++){
    std::cout<<arrayOfElements[i].index<<std::endl;

  }
  std::cout<<"___________________________________"<<std::endl;


  delete [] cputest;
  hipFree(testin);
  hipFree(testout);

  delete [] arrayOfElements;
  hipFree(cudaArrayA);
  hipFree(cudaArrayB);
}

DataStream::DataStream(int numElements, dataPacket * data){
  m_data = data;
  m_numElementsAlive = numElements;
  m_numElements = numElements;

  m_indices = new int[numElements];
  for (int i=0; i<numElements; i+=1){
    m_indices[i] = 1;
  }

  //cudaInit (cudaDataA, cudaDataB, cudaIndicesA, cudaIndicesB);
  hipMalloc ((void**)&cudaDataA, numElements*sizeof (dataPacket));
  hipMalloc ((void**)&cudaDataB, numElements*sizeof (dataPacket));
  hipMalloc ((void**)&cudaIndicesA, numElements*sizeof (int));
  hipMalloc ((void**)&cudaIndicesB, numElements*sizeof (int));

  hipMemcpy(cudaDataA, m_data, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);
  hipMemcpy(cudaDataB, m_data, numElements*sizeof(dataPacket), hipMemcpyHostToDevice);
  hipMemcpy(cudaIndicesA, m_indices, numElements*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudaIndicesB, m_indices, numElements*sizeof(int), hipMemcpyHostToDevice);
}

DataStream::~DataStream(){
  hipFree (cudaDataA);
  hipFree (cudaDataB);
  hipFree (cudaIndicesA);
  hipFree (cudaIndicesB);

  delete [] m_data;
  delete [] m_indices;
}

void DataStream::compact(){

  int numElements = m_numElementsAlive;
  int threadsPerBlock = 64;
  int procsPefBlock = threadsPerBlock*2;

  dim3 initialScanThreadsPerBlock(threadsPerBlock/2);
  dim3 initialScanBlocksPerGrid(numElements/threadsPerBlock);

  dim3 threadsPerBlockL(threadsPerBlock);
  dim3 fullBlocksPerGridL(int(ceil(float(m_numElementsAlive)/float(threadsPerBlock))));

  // // scan algorithm
  //   for (int d=1; d<=ceil(log(m_numElementsAlive)/log(2)); d++){
  //     sum<<<fullBlocksPerGridL, threadsPerBlockL>>>(cudaIndicesA, cudaIndicesB, m_numElementsAlive, powf(2.0f, d-1));
  //     int* temp = cudaIndicesA;
  //     cudaIndicesA = cudaIndicesB;
  //     cudaIndicesB = temp;
  //   }

    test<<<fullBlocksPerGridL, threadsPerBlockL, m_numElementsAlive*sizeof(int)>>>(cudaIndicesA, cudaIndicesB, m_numElementsAlive);
    checkCUDAError("kernel failed!");
    hipMemcpy(m_indices, cudaIndicesB, m_numElements*sizeof(int), hipMemcpyDeviceToHost);
    // //Stream compation from A into B, then save back into A
    // streamCompaction<<<fullBlocksPerGridL, threadsPerBlockL>>>(cudaDataA, cudaIndicesA, cudaDataB, m_numElementsAlive);
    // dataPacket * temp = cudaDataA;
    // cudaDataA = cudaDataB;
    // cudaDataB = temp;

    // // update numrays
    // hipMemcpy(&m_numElementsAlive, &cudaIndicesA[m_numElementsAlive-1], sizeof(int), hipMemcpyDeviceToHost);
}

bool DataStream::getData(int index, dataPacket& data){

  if (index > m_numElements) return false;

  data = m_data[index];
  return true;
}

int DataStream::numAlive(){
  return m_numElementsAlive;
}

void DataStream::fetchDataFromGPU(){
  hipMemcpy(m_data, cudaDataA, m_numElementsAlive*sizeof(dataPacket), hipMemcpyDeviceToHost);
}

void DataStream::kill(int index){
  if (index > m_numElementsAlive) return;

  dim3 threadsPerBlockL(64);
  dim3 fullBlocksPerGridL(int(ceil(float(m_numElementsAlive)/64.0f)));

  killStream<<<fullBlocksPerGridL, threadsPerBlockL>>>(index, cudaDataA, cudaIndicesA, m_numElementsAlive);
}